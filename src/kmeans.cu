#include "hip/hip_runtime.h"
#include "kmeans.h"

template<typename T>
__global__ void get_cluster(T *xdata,
			    T *xcentroids,
			    T *ydata,
			    T *ycentroids,
			    size_t c,
			    size_t *cluster,
			    size_t N)
{
  size_t idx = threadIdx.x + blockIdx.x*blockDim.x;

  if(idx < N)
  {
    float new_dist, dist, distx, disty;
    distx = xdata[idx]-xcentroids[0];
    disty = ydata[idx]-ycentroids[0];
    dist = sqrtf( distx*distx +  disty*disty);
    size_t cl = 0;
  
    for(size_t i = 1; i < c; ++i)
    {
      distx = xdata[idx]-xcentroids[i];
      disty = ydata[idx]-ycentroids[i];
      new_dist = sqrtf( distx*distx +  disty*disty);
      if(new_dist < dist)
      {
        dist = new_dist;
        cl = i;
      }
      __syncthreads();
    }
    
    printf("%ld %ld\n", idx, cl);
    cluster[idx] = cl;
  }
}

__global__ void get_cluster(double *xdata,
			    double *xcentroids,
			    double *ydata,
			    double *ycentroids,
			    size_t c,
			    size_t *cluster,
			    size_t N)
{
  size_t idx = threadIdx.x + blockIdx.x*blockDim.x;
  
  if(idx < N)
  {
    double new_dist, dist, distx, disty;
    distx = xdata[idx]-xcentroids[0];
    disty = ydata[idx]-ycentroids[0];
    dist = sqrt( distx*distx +  disty*disty);
    size_t cl = 0;
  
    for(size_t i = 1; i < c; ++i)
    {
      distx = xdata[idx]-xcentroids[i];
      disty = ydata[idx]-ycentroids[i];
      new_dist = sqrt( distx*distx +  disty*disty);
      if(new_dist < dist)
      {
        dist = new_dist;
        cl = i;
      }
      __syncthreads();
    }

    printf("%ld %ld", idx, cl);
    cluster[idx] = cl;
  }
}

template<typename T>
bool contains(thrust::host_vector<T> &rand,
	      T r,
	      T size)
{
  for(size_t i = 0; i < size; ++i)
  {
    if(rand[i] == r) return true;
  }
  return false;
}

template<typename T>
int read_data(std::string filename,
	      thrust::host_vector<T> &xdata,
	      thrust::host_vector<T> &ydata)
{
  std::ifstream input(filename);
  if(!input)
  {
    std::cerr << "ERROR: could not read " << filename << ".\n";
    return EXIT_FAILURE;
  }
 
  std::string word;
  size_t count = 0;
  while(input >> word)
  {
    if(count == 0)
    {
      xdata.push_back((T)std::stod(word));
      count++;
    }
    else
    {
      ydata.push_back((T)std::stod(word));
      count = 0;
    }
  }
  
  return EXIT_SUCCESS;
}

template<typename T>
void print_vector(thrust::device_vector<T> &v)
{
  for(size_t i = 0; i < v.size(); ++i)
    std::cout << v[i] << " ";
  std::cout << "\n";
}

template<typename T>
void print_vector(thrust::host_vector<T> &v)
{
  for(size_t i = 0; i < v.size(); ++i)
    std::cout << v[i] << " ";
  std::cout << "\n";
}

template<typename T>
size_t kmeans(thrust::host_vector<T> &h_xdata,
	      thrust::host_vector<T> &h_ydata,
	      thrust::host_vector<T> &h_xcentroids,
	      thrust::host_vector<T> &h_ycentroids,
	      size_t K,
	      size_t maxiter)
{
  size_t N = h_xdata.size();
  size_t threadsPerBlock = 256;
  size_t blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
  
  thrust::device_vector<T> d_xdata;
  thrust::device_vector<T> d_ydata;
  d_xdata = h_xdata;
  d_ydata = h_ydata;
  
  thrust::device_vector<T> d_xcentroids(K);
  thrust::device_vector<T> d_ycentroids(K);
  thrust::device_vector<size_t> d_cluster(N);
  
  size_t *offs = (size_t*)malloc(K*sizeof(size_t));
  size_t sum;

  size_t iter = 0;
  bool done = false;
  float resultx, resulty;
  while(iter < maxiter && !done)
  {
    get_cluster<<<blocks,threadsPerBlock>>>
      (thrust::raw_pointer_cast(d_xdata.data()),
       thrust::raw_pointer_cast(d_xcentroids.data()),
       thrust::raw_pointer_cast(d_ydata.data()),
       thrust::raw_pointer_cast(d_ycentroids.data()),
       K,
       thrust::raw_pointer_cast(d_cluster.data()),
       N);
    if(hipDeviceSynchronize() != hipSuccess) return EXIT_FAILURE;
    print_vector(d_cluster);
    
    thrust::device_vector<size_t> help = d_cluster;
    thrust::sort_by_key(thrust::device, d_cluster.begin(), d_cluster.end(), d_xdata.begin());
    thrust::sort_by_key(thrust::device, help.begin(), help.end(), d_ydata.begin());
    help.resize(0);
    
    std::cout << "cluster offsets:\n";
    sum = 0;
    for(size_t i = 0; i < K; ++i)
    {
      offs[i] = sum;
      std::cout << sum << " ";
      sum += thrust::count(d_cluster.begin(), d_cluster.end(), i);
    }
    std::cout << "\n";
    
    done = true;
    for(size_t i = 0; i < K; ++i)
    {
      if(i == K-1)
      {
	resultx = thrust::reduce(thrust::device,
				 thrust::raw_pointer_cast(d_xdata.data())+offs[K-1],
				 thrust::raw_pointer_cast(d_xdata.data())+N,
				 0);
	resulty = thrust::reduce(thrust::device,
				 thrust::raw_pointer_cast(d_xdata.data())+offs[K-1],
				 thrust::raw_pointer_cast(d_xdata.data())+N,
				 0);
      }
      else
      {
	resultx = thrust::reduce(thrust::device,
				 thrust::raw_pointer_cast(d_xdata.data())+offs[i],
				 thrust::raw_pointer_cast(d_xdata.data())+offs[i+1],
				 0);
	resulty = thrust::reduce(thrust::device,
				 thrust::raw_pointer_cast(d_xdata.data())+offs[i],
				 thrust::raw_pointer_cast(d_xdata.data())+offs[i+1],
				 0);
      }

      if(d_xcentroids[i] != resultx || d_ycentroids[i] != resulty) done = false;
      d_xcentroids[i] = resultx;
      d_ycentroids[i] = resulty;
    }

    iter++;
  }
  
  h_xdata = d_xdata;
  h_ydata = d_ydata;
  h_xcentroids = d_xcentroids;
  h_ycentroids = d_ycentroids;

  return iter;
}

template<typename T>
int seeding(thrust::host_vector<T> &h_xdata,
	    thrust::host_vector<T> &h_ydata,
	    thrust::host_vector<T> &h_xcentroids,
	    thrust::host_vector<T> &h_ycentroids)
{
  srand(time(NULL));

  size_t K = h_xcentroids.size();
  size_t N = h_xdata.size();
  if(K > N)
  {
    std::cerr << "More clusters than data points!\n";
    return EXIT_FAILURE;
  }
    
  thrust::host_vector<size_t> randv(K);
  size_t r;
  std::cout << "initial values:\n";
  for(size_t i = 0; i < K; ++i)
  {
    while(contains(randv,r,K)) r = rand()%N;
    randv[i] = r;
    h_xcentroids[i] = h_xdata[r];
    h_ycentroids[i] = h_ydata[r];
    std::cout << h_xcentroids[i] << " " << h_ycentroids[i] << "\n";
  }
  std::cout << "\n";

  return EXIT_SUCCESS;
}
